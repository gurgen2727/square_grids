//Header from standard libraries

#include <hip/hip_runtime.h>
#include <fstream>

//size of blocks in grid
#define BLOCK_SIZE 16

//kernel function  to multiply c=a*b
__global__ void Muld(float* a,float* b,float* c,int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	float sum = 0;

	if(i>=n || j>=n){
		return;
	}

	for(int k = 0 ; k < n ; ++k) {
		sum = sum + a[i*n + k]*b[k*n + j];
	}

	c[i*n+j] = sum;
}

//host function  to multiply C=A*B
void Mul(float* A, float* B, int n,float* C)
{
        int size;
        // Load A and B to the device
        float* Ad;
        size = n * n * sizeof(float);
        hipMalloc((void**)&Ad, size);
        hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
        float* Bd;
        size = n * n * sizeof(float);
        hipMalloc((void**)&Bd, size);
        hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
        // Allocate C on the device
        float* Cd;
        size = n * n * sizeof(float);
        hipMalloc((void**)&Cd, size);
        // Compute the execution configuration assuming
        // the matrix dimensions are multiples of BLOCK_SIZE
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid(n / dimBlock.x + (n%dimBlock.x!=0), n / dimBlock.y+(n%dimBlock.y!=0));
        // Launch the device computation
        Muld<<<dimGrid, dimBlock>>>(Ad, Bd, Cd , n);
        // Read C from the device
        hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
        // Free device memory
        hipFree(Ad);
        hipFree(Bd);
        hipFree(Cd);
}

int main(int argc,char* argv[])
{
	std::ifstream  fin(argv[1]);
	std::ofstream fout("out.txt");
	int n;
	fin >> n;
        float *hA = (float*)malloc(sizeof(float)*n*n);
        float *hB = (float*)malloc(sizeof(float)*n*n);
        for(int i = 0; i < n; ++i) {
                for(int j = 0; j < n; ++j) {
                        fin >> hA[i*n+j];
                }
        }
        for(int i = 0; i < n; ++i) {
                for(int j = 0; j < n; ++j) {
                        hB[i*n+j] = 0;
                }
		hB[i*n+i] = 1;
        }
	// in case, when count of input matrix > 1
	// hC = hA * hB;
	// Mul(hA,hB,n,hC);
	for(int i = 2; i <= n; i <<=1) {
		Mul(hA,hA,n,hA);
		if(i & n) {
			Mul(hA,hB,n,hB);
		}
	}
        for(int i = 0; i < n; ++i) {
                for(int j = 0; j < n; ++j) {
                        fout << hB[i*n+j] << ' ';
		}
		fout << std::endl;
        }
	free(hA);
	free(hB);
        return 0;
}

